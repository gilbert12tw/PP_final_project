#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024
#define CHUNK_SIZE 4096
#define max(a, b) (a > b ? a : b)
#define INF ((1 << 30) - 1)

struct Item {
    int weight, value;
};

static inline bool compareByValue(const Item &a, const Item &b) {
    if (a.value == b.value) {
        return a.weight < b.weight;
    }
    return a.value < b.value;
}


static inline bool compareByWeight(const Item &a, const Item &b) {
    if (a.weight == b.weight) {
        return a.value > b.value;
    }
    return a.weight < b.weight;
}

static inline int ceil_div(int a, int b) { return (a + b - 1) / b; }

__global__ void mckp_kernel(int *dp_prev, int *dp_curr, int *group_values, int group_weights) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int shared_values[CHUNK_SIZE];

#pragma unroll(4096 / 1024)
    for (int i = threadIdx.x; i < CHUNK_SIZE; i += blockDim.x)
        shared_values[i] = group_values[i];
    __syncthreads();

    int maxVal = dp_prev[idx];
    int sumV = 0;

    for (int k = 0; k < CHUNK_SIZE; k++) {
        sumV += shared_values[k];
        if (group_weights * (k + 1) > idx) break;

        maxVal = max(maxVal, dp_prev[idx - group_weights * (k + 1)] + sumV);
    }

    dp_curr[idx] = maxVal;
}

void input(char *infile, int &n, int &m, Item *&items) {
    FILE *file = fopen(infile, "rb");
    if (!file) {
        printf("Error: Cannot open input file\n");
        exit(1);
    }

    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    items = (Item *)malloc(n * sizeof(Item));

    fread(items, sizeof(int), 2 * n, file);

    fclose(file);
}

void processGroups(Item *items, int n, int *&group_counts, int *&unique_weight, int &num_groups) {
    std::sort(items, items + n, compareByWeight);

    num_groups = 1;
    int curr_weight = items[0].weight;
    int g_cnt = 1;
    for (int i = 1; i < n; i++) {
        if (items[i].weight != curr_weight || g_cnt >= CHUNK_SIZE) {
            num_groups++;
            curr_weight = items[i].weight;
            g_cnt = 1;
        } else {
            ++g_cnt;
        }
    }

    group_counts = (int *)malloc(num_groups * sizeof(int));
    unique_weight = (int *)malloc(num_groups * sizeof(int));
    memset(group_counts, 0, num_groups * sizeof(int));

    int group_idx = 0;
    curr_weight = items[0].weight;
    group_counts[0] = 1;
    unique_weight[0] = curr_weight;

    for (int i = 1; i < n; i++) {
        if (items[i].weight != curr_weight || group_counts[group_idx] >= CHUNK_SIZE) {
            group_idx++;
            curr_weight = items[i].weight;
            unique_weight[group_idx] = curr_weight;
            group_counts[group_idx] = 1;
        } else {
            group_counts[group_idx]++;
        }
    }
}

void output(char *outFileName, int result, int m) {
    FILE *outfile = fopen(outFileName, "w");
    if (!outfile) {
        printf("Error: Cannot open output file\n");
        exit(1);
    }

    fwrite(&result, sizeof(int), 1, outfile);
    fclose(outfile);
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_file> <output_file>\n", argv[0]);
        return 1;
    }

    int n, m;
    Item *items = NULL;

    input(argv[1], n, m, items);

    int *group_counts = NULL;
    int *unique_weight = NULL;
    int num_groups;

    processGroups(items, n, group_counts, unique_weight, num_groups);

    int const m_pad = ceil_div(m + 1, CHUNK_SIZE) * CHUNK_SIZE;
    int *d_dp_prev = NULL, *d_dp_curr = NULL;
    hipMalloc((void **)&d_dp_prev, m_pad * sizeof(int));
    hipMalloc((void **)&d_dp_curr, m_pad * sizeof(int));

    // cudaMemset(d_dp_prev, 0, m_pad * sizeof(int));
    // cudaMemset(d_dp_curr, 0, m_pad * sizeof(int));

    int numBlocks = m_pad / BLOCK_SIZE;

    int curr_pos = 0;
    int *d_group_values = NULL;
    hipMalloc((void **)&d_group_values, CHUNK_SIZE * sizeof(int));
    int *group_values = NULL;
    hipHostMalloc(&group_values, CHUNK_SIZE * sizeof(int), hipHostMallocDefault);
    printf("num_groups = %d\n", num_groups);
    for (int g = 0; g < num_groups; g++) {
        #pragma omp parallel for
        for (int i = 0; i < CHUNK_SIZE; i++)
            group_values[i] = (i < group_counts[g] ? items[curr_pos + i].value : 0);

        hipMemcpy(d_group_values, group_values,
                   CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice);

        mckp_kernel<<<numBlocks, BLOCK_SIZE>>>(d_dp_prev, d_dp_curr, d_group_values, unique_weight[g]);

        int *temp = d_dp_prev;
        d_dp_prev = d_dp_curr;
        d_dp_curr = temp;

        curr_pos += group_counts[g];
    }
    hipFree(d_group_values);
    hipHostFree(group_values);

    int result;
    hipMemcpy(&result, &d_dp_prev[m], sizeof(int), hipMemcpyDeviceToHost);

    output(argv[2], result, m);

    hipFree(d_dp_prev);
    hipFree(d_dp_curr);
    free(group_counts);
    free(unique_weight);
    free(items);

    return 0;
}
