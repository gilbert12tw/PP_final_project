#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 1024
#define CHUNK_SIZE 4096

#define max(a, b) (a > b ? a : b)
#define INF ((1 << 30) - 1)

struct Item {
    int weight, value;
};

static inline bool compareByValue(const Item &a, const Item &b) {
    if (a.value == b.value) {
        return a.weight < b.weight;
    }
    return a.value < b.value;
}
static inline int ceil_div(int a, int b) { return (a + b - 1) / b; }

__global__ void mckp_kernel(int *dp_prev, int *dp_curr,
                            int *group_weights, int group_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ int shared_weights[CHUNK_SIZE];

#pragma unroll(4096 / 1024)
    for (int i = threadIdx.x; i < CHUNK_SIZE; i += blockDim.x)
        shared_weights[i] = group_weights[i];
    __syncthreads();

    int maxVal = dp_prev[idx];
    int sumW = 0;

    for (int k = 0; k < CHUNK_SIZE; k++) {
        sumW += shared_weights[k];
        if (sumW > idx) break;

        maxVal = max(maxVal, dp_prev[idx - sumW] + group_value * (k + 1));
    }

    dp_curr[idx] = maxVal;
}

void input(char *infile, int &n, int &m, Item *&items) {
    FILE *file = fopen(infile, "rb");
    if (!file) {
        printf("Error: Cannot open input file\n");
        exit(1);
    }

    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    items = (Item *)malloc(n * sizeof(Item));

    fread(items, sizeof(int), 2 * n, file);

    fclose(file);
}

void processGroups(Item *items, int n, int *&group_counts, int *&unique_values, int &num_groups) {
    std::sort(items, items + n, compareByValue);

    num_groups = 1;
    int curr_value = items[0].value;
    int g_cnt = 1;
    for (int i = 1; i < n; i++) {
        if (items[i].value != curr_value || g_cnt >= CHUNK_SIZE) {
            num_groups++;
            curr_value = items[i].value;
            g_cnt = 1;
        } else {
            ++g_cnt;
        }
    }

    group_counts = (int *)malloc(num_groups * sizeof(int));
    unique_values = (int *)malloc(num_groups * sizeof(int));
    memset(group_counts, 0, num_groups * sizeof(int));

    int group_idx = 0;
    curr_value = items[0].value;
    group_counts[0] = 1;
    unique_values[0] = curr_value;

    for (int i = 1; i < n; i++) {
        if (items[i].value != curr_value || group_counts[group_idx] >= CHUNK_SIZE) {
            group_idx++;
            curr_value = items[i].value;
            unique_values[group_idx] = curr_value;
            group_counts[group_idx] = 1;
        } else {
            group_counts[group_idx]++;
        }
    }
}

void output(char *outFileName, int result, int m) {
    FILE *outfile = fopen(outFileName, "w");
    if (!outfile) {
        printf("Error: Cannot open output file\n");
        exit(1);
    }

    fwrite(&result, sizeof(int), 1, outfile);
    fclose(outfile);
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        printf("Usage: %s <input_file> <output_file>\n", argv[0]);
        return 1;
    }

    int n, m;
    Item *items = NULL;

    input(argv[1], n, m, items);

    int *group_counts = NULL;
    int *unique_values = NULL;
    int num_groups;

    processGroups(items, n, group_counts, unique_values, num_groups);

    int const m_pad = ceil_div(m + 1, CHUNK_SIZE) * CHUNK_SIZE;
    int *d_dp_prev = NULL, *d_dp_curr = NULL;
    hipMalloc((void **)&d_dp_prev, m_pad * sizeof(int));
    hipMalloc((void **)&d_dp_curr, m_pad * sizeof(int));

    // cudaMemset(d_dp_prev, 0, m_pad * sizeof(int));
    // cudaMemset(d_dp_curr, 0, m_pad * sizeof(int));

    int numBlocks = m_pad / BLOCK_SIZE;

    int curr_pos = 0;
    int *d_group_weights = NULL;
    hipMalloc((void **)&d_group_weights, CHUNK_SIZE * sizeof(int));
    int *group_weights = (int*)malloc(CHUNK_SIZE * sizeof(int));
    for (int g = 0; g < num_groups; g++) {
#pragma omp parallel for
        for (int i = 0; i < CHUNK_SIZE; i++)
            group_weights[i] = (i < group_counts[g] ? items[curr_pos + i].weight : INF);

        hipMemcpy(d_group_weights, group_weights,
                   CHUNK_SIZE * sizeof(int), hipMemcpyHostToDevice);

        mckp_kernel<<<numBlocks, BLOCK_SIZE>>>(d_dp_prev, d_dp_curr,
                                               d_group_weights, unique_values[g]);

        int *temp = d_dp_prev;
        d_dp_prev = d_dp_curr;
        d_dp_curr = temp;

        curr_pos += group_counts[g];
    }
    hipFree(d_group_weights);
    free(group_weights);

    int result;
    hipMemcpy(&result, &d_dp_prev[m], sizeof(int), hipMemcpyDeviceToHost);

    output(argv[2], result, m);

    hipFree(d_dp_prev);
    hipFree(d_dp_curr);
    free(group_counts);
    free(unique_values);
    free(items);

    return 0;
}
